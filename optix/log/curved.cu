#include "hip/hip_runtime.h"
/***************************************************************/
//
//Copyright (c) 2019 Esteban Egea-Lopez http://girtel.upct.es/~simulations
//
/**************************************************************/

//License from NVIDIA parts
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "../../Common.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
using namespace optix;

//Launch variables
rtDeclareVariable(uint3, launchIndexTriangle, rtLaunchIndex, );
rtDeclareVariable(CurvedMeshLPWavePayload, rayPayload, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(CurvedTriangleHit, ch_triangle_data, attribute curved_triangle_hit_data, );


RT_PROGRAM void closestHitCurvedLogTrace()
{

	//Update payload
	//Get the hitpoint from the barycentric coordinates computed in the triangle hit. This should get us a point always on the surface and help avoid self-intersection
	//See https://www.realtimerendering.com/raytracinggems/ 6.1
	const float3 lastHP=make_float3(rayPayload.hitPointAtt.x,rayPayload.hitPointAtt.y,rayPayload.hitPointAtt.z);
	//we could use t of ray, but if we shift the ray over the normal to avoid self-intersection we introduce an error in the electric field
       
	 //Segment length from last physical interaction to this physical interaction
	const float3 lastReflectionHitPoint = make_float3(rayPayload.lrhpd.x,rayPayload.lrhpd.y,rayPayload.lrhpd.z);

	//The rayLength is the distance from the last hit point to this hit point. Notice that, for instance, if the last hp is on a sphere receiver, there was no actual physical interaction with 
        //any element. In that case, this rayLength is different from s_prime. It is only used to update the total distance (unfolded) of this ray. 
	//TODO: it is used in the basic setup, for curved may be removed
	const float rayLength=length(ch_triangle_data.hp-lastHP);
	rayPayload.hitPointAtt.x =ch_triangle_data.hp.x;
	rayPayload.hitPointAtt.y =ch_triangle_data.hp.y;
	rayPayload.hitPointAtt.z =ch_triangle_data.hp.z;
	//rtPrintf("THP\t%u\t%u\tbary=(%.6e,%.6e,%.6e)\n",launchIndexTriangle.x,launchIndexTriangle.y,ch_triangle_data.hp.x,ch_triangle_data.hp.y,ch_triangle_data.hp.z);
	const float3 gn=make_float3(ch_triangle_data.geom_normal_t.x,ch_triangle_data.geom_normal_t.y,ch_triangle_data.geom_normal_t.z);	
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,gn )); //Plane normal
	
	//Redundant
	//if (curvedMesh==1u) {
	rayPayload.rhfr.z |= 1u<<FLAG_CURVED_MESH_POSITION;
	//rayPayload.rhfr.z=rayPayload.rhfr.z+FLAG_CURVED_MESH;
	//}

#ifdef OPAL_AVOID_SI
	rayPayload.lastNormal=n;
#endif	
	const float3 reflection_dir=normalize(reflect(ray.direction, n));
	const float aux=rayPayload.ndtd.w; //previous total distance of the ray
	rayPayload.ndtd = make_float4(reflection_dir); //initialized with float3, w is set to 0. and updated below
	
	//Use reflections and hits to create hash
	
	//hash_combine_impl<uint>(rayPayload.refhash,ch_triangle_data.faceId+rayPayload.reflections+rayPayload.hits);
	rayPayload.ndtd.w = aux+ rayLength; //Update total distance of the ray
	
	
	
	//++rayPayload.reflections;
	uint reflections=rayPayload.rhfr.x;
	rayPayload.rhfr.x=reflections+1u;

}




