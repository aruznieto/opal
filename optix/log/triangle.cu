#include "hip/hip_runtime.h"
/***************************************************************/
//
//Copyright (c) 2019 Esteban Egea-Lopez http://girtel.upct.es/~simulations
//
/**************************************************************/

//License from NVIDIA parts
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "../../Common.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
//#include <cmath>
using namespace optix;

//Launch variables
rtDeclareVariable(uint3, launchIndexTriangle, rtLaunchIndex, );
rtDeclareVariable(BaseReflectionPayload, rayPayload, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(TriangleHit, ch_triangle_data, attribute triangle_hit_data, );

//Per-mesh local variables 
rtDeclareVariable(uint, meshId, , );


//Only for reflections, penetrations are not visualized yet
RT_PROGRAM void closestHitTriangleLogTrace()
{

	//Update payload
	//const float rayLength = ch_triangle_data.geom_normal_t.w;
	//const float3 hp= ray.origin + rayLength * ray.direction ;
	//rayPayload.hitPoint =hp;
	//Get the hitpoint from the barycentric coordinates computed in the triangle hit. This should get us a point always on the surface and help avoid self-intersection
	//See https://www.realtimerendering.com/raytracinggems/ 6.1
	const float3 lastHP=make_float3(rayPayload.hitPointAtt.x,rayPayload.hitPointAtt.y,rayPayload.hitPointAtt.z);
	//we could use t of ray, but if we shift the ray over the normal to avoid self-intersection we introduce an error in the electric field
	const float rayLength=length(ch_triangle_data.hp-lastHP);
	rayPayload.hitPointAtt.x =ch_triangle_data.hp.x;
	rayPayload.hitPointAtt.y =ch_triangle_data.hp.y;
	rayPayload.hitPointAtt.z =ch_triangle_data.hp.z;
	const float3 gn=make_float3(ch_triangle_data.geom_normal_t.x,ch_triangle_data.geom_normal_t.y,ch_triangle_data.geom_normal_t.z);	
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,gn )); //Plane normal
	const float3 reflection_dir=reflect(ray.direction, n);
	const float aux=rayPayload.ndtd.w;
	rayPayload.ndtd = make_float4(reflection_dir); //initialized with float3, w is set to 0. and updated below
	rayPayload.ndtd.w = aux+ rayLength;
#ifdef OPAL_AVOID_SI
	rayPayload.lastNormal=n;
#endif	
	
	
	uint reflections=rayPayload.rhfr.x;
	rayPayload.rhfr.x=reflections+1u;
	

}



