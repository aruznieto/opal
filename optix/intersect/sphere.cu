#include "hip/hip_runtime.h"
/***************************************************************/
//
//Copyright (c) 2019 Esteban Egea-Lopez http://girtel.upct.es/~simulations
//
/**************************************************************/


//Modified from NVIDIA OptiX samples

/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "../../Common.h"
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
using namespace optix;





//Sphere variables
rtDeclareVariable(float4, sphere, , );
rtDeclareVariable(SphereHit, hit_attr, attribute hit_attr, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

template<bool use_robust_method>
static __device__
void intersect_sphere(void)
{
	float3 center = make_float3(sphere);
	float3 O = ray.origin - center;
	float3 D = ray.direction;
	float radius = sphere.w;

	float b = dot(O, D);
	float c = dot(O, O) - radius*radius;
	float disc = b*b - c;
	if (disc > 0.0f) {
		float sdisc = sqrtf(disc);
		float root1 = (-b - sdisc);

		bool do_refine = false;

		float root11 = 0.0f;

		if (use_robust_method && fabsf(root1) > 10.f * radius) {
			do_refine = true;
		}

		if (do_refine) {
			// refine root1
			float3 O1 = O + root1 * ray.direction;
			b = dot(O1, D);
			c = dot(O1, O1) - radius*radius;
			disc = b*b - c;

			if (disc > 0.0f) {
				sdisc = sqrtf(disc);
				root11 = (-b - sdisc);
			}
		}

		bool check_second = true;
		float t = root1 + root11;
		if (rtPotentialIntersection(t)) {
			//shading_normal = geometric_normal = (O + (root1 + root11)*D) / radius;
			SphereHit h;
//			h.t = t;
//			h.geom_normal = (O + (root1 + root11)*D) / radius;
		float3 gn = (O + (root1 + root11)*D) / radius;		
	//packed version
			h.geom_normal_t=make_float4(gn.x,gn.y,gn.z,t);

			hit_attr = h;
			//Only one material. Change here if more materials used
			if (rtReportIntersection(0)) {
				check_second = false;
			}
		}
		if (check_second) {
			float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
			if (rtPotentialIntersection(root2)) {
				SphereHit h;
				//h.t = root2;
				//h.geom_normal = (O + root2*D) / radius;
				float3 gn2 = (O + root2*D) / radius;
				//Packed version
				h.geom_normal_t=make_float4(gn2.x,gn2.y,gn2.z,root2);
				hit_attr = h;
				//shading_normal = geometric_normal = (O + root2*D) / radius;
			
				//Only one material. Change here if more materials used
				rtReportIntersection(0);
			}
		}
	}
}


RT_PROGRAM void intersectSphere(int primIdx)
{
	intersect_sphere<false>();
}


RT_PROGRAM void robust_intersectSphere(int primIdx)
{
	intersect_sphere<true>();
}
RT_PROGRAM void rtgem_intersectSphere(int primIdx) {
	float3 center = make_float3(sphere);
	float3 f = ray.origin - center;
	float3 d = ray.direction; 
	float radius = sphere.w;
	//float a=dot(d,d); Removed because we assume the ray direction is already normalized

	float b=-1.0f*dot(f,d);
	float3 aux=f+(b*d);
	float rsq=radius*radius;
	float disc=rsq - dot(aux,aux);
	if (disc > 0.0f) { //A tangential hit is not considered a hit
		float c=dot(f,f)-rsq;
		float s=((b > 0) ? 1.0f : -1.0f); 
		float q=b+(s*sqrt(disc));
		float t=c/q;
		bool check_second = true;
		if (rtPotentialIntersection(t)) {
			//rtPrintf("primIdx=%d,s=%f,q=%f,t=%f\n",primIdx,s,q,t);
			//shading_normal = geometric_normal = (O + (root1 + root11)*D) / radius;
			SphereHit h;
//			h.t = t;
//			h.geom_normal = (O + (root1 + root11)*D) / radius;
			float3 gn = (f + (t*d)) / radius;		
			//packed version
			h.geom_normal_t=make_float4(gn.x,gn.y,gn.z,t);

			hit_attr = h;
			//Only one material. Change here if more materials used
			if (rtReportIntersection(0)) {
				check_second = false;
			}
		}
		if (check_second) {
			t=q;	
			if (rtPotentialIntersection(t)) {
				SphereHit h;
				//rtPrintf("primIdx=%d,2s=%f,q=%f,t=%f\n",primIdx,s,q,t);
				float3 gn2 = (f + (t*d)) / radius;		
				//Packed version
				h.geom_normal_t=make_float4(gn2.x,gn2.y,gn2.z,t);
				hit_attr = h;
				//Only one material. Change here if more materials used
				rtReportIntersection(0);
			}
		}
	}

}


RT_PROGRAM void boundsSphere(int, float result[6])
{
	const float3 cen = make_float3(sphere);
	const float3 rad = make_float3(sphere.w);

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (rad.x > 0.0f && !isinf(rad.x)) {
		aabb->m_min = cen - rad;
		aabb->m_max = cen + rad;
	}
	else {
		aabb->invalidate();
	}
}

