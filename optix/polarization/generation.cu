#include "hip/hip_runtime.h"
/***************************************************************/
//
//Copyright (c) 2019 Esteban Egea-Lopez http://girtel.upct.es/~simulations
//
/**************************************************************/


#include "../../Common.h"
#include "../traceFunctions.h"
#include "../configuration.h"
#include "../receiverFunctions.h" //For antenna gain
#include "linearPolarizationFunctions.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
using namespace optix;


//Generation of ray sphere


//Ray Sphere buffer
rtBuffer<float3, 2> raySphere2D;

//Ray range buffer
//rtBuffer<float4, 1> rayRangeBuffer;


//Transmitter buffer
rtBuffer<Transmitter, 1> txBuffer;

//Sphere parameters buffer
rtBuffer<uint4, 1> raySphereParametersBuffer;
//Launch variables
rtDeclareVariable(uint3, launchIndex, rtLaunchIndex, );

rtDeclareVariable(uint, rayTypeIndex, , );


//Configuration variables
//rtDeclareVariable(uint2, raySphereSize, , );
//rtDeclareVariable(uint, usePenetration, , );
rtDeclareVariable(uint, initialHash, , );
//rtDeclareVariable(uint, standardSphere, , );



RT_PROGRAM void genRayAndReflectionsFromSphereIndex()
{


	//3D kernel launch [elevation, azimuth, transmitters]	

	uint2 idx = make_uint2(launchIndex.x, launchIndex.y); //[elevation, azimuth]
	const uint standardSphere= raySphereParametersBuffer[0].z;
	const uint2 raySphereSize = make_uint2(raySphereParametersBuffer[0].x,raySphereParametersBuffer[0].y);
	if (standardSphere==1u) {
		//index goes from 0 to raySphereSize.x-1 //The last elevation step corresponds to 180 degrees elevation
		if ((idx.x == 0 ||idx.x==  raySphereSize.x-1  ) && idx.y != 0) {
			//These rays are all the same (0,1,0) or (0,-1,0). Only trace  (0,0) and (last,0) corresponding to 0 and 180 elevation degrees
			return;
		}
	}
	float3 ray_direction = raySphere2D[idx];

	const Transmitter tx = txBuffer[launchIndex.z];
	
	float3 origin = make_float3(tx.origin_p);
	

	LPWavePayload rayPayload;
	rayPayload.ndtd = optix::make_float4(0.0f);
	//rayPayload.hitPoint = origin;
	rayPayload.hitPointAtt =make_float4(origin);
	rayPayload.hitPointAtt.w=0.0f;

	rayPayload.hor_coeff=make_float2(1.0f,0.0f);	
	rayPayload.ver_coeff=make_float2(1.0f,0.0f);	

	fillPolarization(rayPayload,make_float3(tx.polarization_k), ray_direction);
	
	//rtPrintf("\t%u\t%u\tray=(%f,%f,%f),pol=(%f,%f,%f), polt=(%f,%f,%f)\n",launchIndex.x, launchIndex.y,ray_direction.x,ray_direction.y,ray_direction.z,tx.polarization.x,tx.polarization.y,tx.polarization.z,rayPayload.E.x,rayPayload.E.y,rayPayload.E.z);
	//rtPrintf("G\t%u\t%u\tray=(%f,%f,%f),pol=(%f,%f,%f),\n",launchIndex.x, launchIndex.y,ray_direction.x,ray_direction.y,ray_direction.z,tx.polarization.x,tx.polarization.y,tx.polarization.z);
	//rtPrintf("G\t%u\t%u\thor_v=(%.6e,%.6e,%.6e),ver_v=(%.6e,%.6e,%.6e), hor_coeff=(%.6e,%.6e), ver_coeff(%.6e,%.6e)\n",launchIndex.x, launchIndex.y,rayPayload.hor_v.x,rayPayload.hor_v.y,rayPayload.hor_v.z,rayPayload.ver_v.x,rayPayload.ver_v.y,rayPayload.ver_v.z, rayPayload.hor_coeff.x,rayPayload.hor_coeff.y,rayPayload.ver_coeff.x,rayPayload.ver_coeff.y);
	
	rayPayload.lrhpd = make_float4(origin);
	rayPayload.lrhpd.w = 0.0f; //totalDistanceTillLastReflection
	rayPayload.polarization_k=tx.polarization_k;
	//TODO: Add possibilty of differentInitialFieldAmplitude;	
	if (useAntennaGain) {
		rtBufferId<float,2> bid=tx.gainId;
		const Matrix<4,4> tp=tx.transformToPolarization;
		float g=getAntennaGain(ray_direction,bid,tp) ;	
		rayPayload.electricFieldAmplitude = g; //Gain is already in electric field units, no need to convert from dB or take sqrt 
	} else {
		rayPayload.electricFieldAmplitude = 1.0f; //Normalized Eo=1. Antenna Gain = 1. 
	}
	//rayPayload.accumulatedAttenuation=0.0f;
	rayPayload.rhfr=make_uint4(0u,0u,FLAG_NONE,initialHash);
	rayPayload.initialRayDir=make_float4(ray_direction);

	
//	float2 ang= getAngles(ray_direction);
	//Print all rays generated
//	rtPrintf("A\t%u\t%u\t%f\t%f\t%f\t%f\t%f\n", launchIndex.x, launchIndex.y, (ang.x*180.f/M_PIf), (ang.y*180.f/M_PIf), ray_direction.x, ray_direction.y, ray_direction.z);

	//trace ray
	traceReflection<LPWavePayload>(rayPayload, rayTypeIndex, origin, ray_direction, launchIndex.x,launchIndex.y,false);


}


