#include "hip/hip_runtime.h"
/***************************************************************/
//
//Copyright (c) 2019 Esteban Egea-Lopez http://girtel.upct.es/~simulations
//
/**************************************************************/
#include "../../../Common.h"
#include <optix_world.h>
#include "../linearPolarizationFunctions.h"
#include "../../configuration.h"
#include "../../reflectionFunctions.h"
#include "../../receiverFunctions.h"
#include "diffractionFunctions.h"
using namespace optix;


//Scene root
//rtDeclareVariable(rtObject, root, , ); //Already defined in trace functions
//Static meshes root
//rtDeclareVariable(rtObject, staticMeshesRoot, , );

//Edge buffer
rtBuffer<Edge, 1> edgeBuffer;

//TODO:These  should be redundant if we used Receiver buffers
//Receiver position buffer 
rtBuffer<float4, 1> receiverPositionsBuffer;
rtBuffer<float3, 1> receiverPolarizationBuffer;
rtBuffer<rtBufferId<float,2>, 1> antennaGainIdBuffer;
typedef optix::Matrix<4,4> TransMat; 
rtBuffer<TransMat, 1> transformToPolarizationBuffer;

//Hit buffer
rtBuffer<RDNHit, 1> difBuffer; //Buffer to store all the hits
//Real hits buffer
rtBuffer<uint3, 1> losBuffer; 

//Local variables
//rtDeclareVariable(float, k, , ); //wavenumber 2pi/lambda
rtDeclareVariable(uint, computeMode, ,);
rtDeclareVariable(uint, traceDiffraction, ,);

//Transmitter buffer
rtBuffer<Transmitter, 1> txBuffer;

//Launch variables
rtDeclareVariable(uint, launchIndex, rtLaunchIndex, );

//Visibility ray payload
rtDeclareVariable(VisibilityPayload, rayPayload, rtPayload, );
rtDeclareVariable(uint, rayTypeIndex, , );

//For debug only
rtDeclareVariable(TriangleHit, ch_triangle_data, attribute triangle_hit_data, );
rtDeclareVariable(CurvedTriangleHit, curved_triangle_data, attribute curved_triangle_hit_data, );
rtDeclareVariable(optix::Ray, ray_hit, rtCurrentRay, );


rtBuffer<LogTraceHitInfo, 1> traceBufferDiffraction;
rtBuffer<uint, 1> traceAtomicIndexDiffraction; //Buffer to store the current log trace buffer index 



__forceinline__ __device__ void computeElectricFieldAtReceiver(const float4& angles, const float2& spar, const float3& txRay, const float4& polarization_k, const float3& n_iplane, const float3& n_dplane, float n, float3& rxRay, float4& R_0, float4& R_n , const float gain ) {
	uint3 index=losBuffer[launchIndex];
	const uint edgeIndex=	index.x;
	const uint rxIndex = index.y;
	const uint txIndex = index.z;
	const float k = polarization_k.w;
	const float3 pol=make_float3(polarization_k);
	const float s=spar.x;
	const float s_prime=spar.y;
	//Parameter A for spherical waves, see Balanis TODO: add support for other waves in the future
	const float A=sqrtf(s_prime/(s*(s_prime+s))); 
	const float sinbetap=sinf(angles.y);
	//L parameter for spherical and conical incidence, see Balanis. TODO: add support for other waves in the future 
	const float L=(s*s_prime*sinbetap*sinbetap)/(s_prime+s);
	//Get the electrical field vector for this ray
	const float3 Ev=getLinearPolarizationForRaySimple(pol,txRay); //It is normalized

		//rtPrintf("%u\tray=(%f,%f,%f) pol=(%f,%f,%f) Ev=(%f,%f,%f) A=%f L=%f)\n",launchIndex.y,txRay.x,txRay.y,txRay.z,pol.x,pol.y,pol.z,Ev.x,Ev.y,Ev.z,A,L);
		//rtPrintf("%u\tray=(%f,%f,%f)  Ev=(%f,%f,%f) A=%f L=%f s=%f,s'=%f)\n",launchIndex.x,txRay.x,txRay.y,txRay.z,Ev.x,Ev.y,Ev.z,A,L,s,s_prime);
		//rtPrintf("rt=(%f,%f,%f) pol=(%f,%f,%f) Ev2=(%f,%f,%f) Evsimple2=(%f,%f,%f)\n",rt.x,rt.y,rt.z,pol.x,pol.y,pol.z,Ev3.x,Ev3.y,Ev3.z,Ev4.x,Ev4.y,Ev4.z);

	//Get the unit vector parallel to the plane of incidence
	const float3 phi_p = n_iplane;
	const float3 beta_p=normalize(cross(txRay,phi_p)); 
	const float3 ss=normalize(cross(phi_p,beta_p)); 
		//rtPrintf("%u\t beta_p=(%f,%f,%f) phi_p=(%f,%f,%f) s_p=(%f,%f,%f) \n",launchIndex.x,beta_p.x,beta_p.y,beta_p.z,phi_p.x,phi_p.y,phi_p.z,ss.x,ss.y,ss.z);

	//Compute incident electric field at the point of diffraction (complex)	
	float2 z = make_float2(0.0f, -k*s_prime);
	float2 zexp = complex_exp_only_imaginary(z);
	const float2 Ei = sca_complex_prod((gain/s_prime), zexp); //TODO: Assuming the initial amplitude is 1.0. To Change if antenna radiation patter or similar is used 

	//Geometric parts
	const float Ei_beta=dot(beta_p,Ev); //component of the incident E field parallel to the plane of incidence at the point of diffraction
	const float Ei_phi=dot(phi_p,Ev); //component of the incident E field perpendicular to the plane of incidence at the point of diffraction



	//Incident complex amplitude components
	float2 Ei_beta_q=sca_complex_prod(Ei_beta,Ei);
	float2 Ei_phi_q=sca_complex_prod(Ei_phi,Ei);
		//rtPrintf("%u\t Ev=(%f,%f,%f) Ei_beta=%f Ei_phi=%f Ei_beta_q=(%f,%f) Ei_phi_q=(%f,%f) an=%f\n",launchIndex.y,Ev.x,Ev.y,Ev.z,Ei_beta,Ei_phi, Ei_beta_q.x,Ei_beta_q.y, Ei_phi_q.x,Ei_phi_q.y,(atan2(Ei_phi_q.y,Ei_phi_q.x)*180/M_PIf));


	//Diffraction coefficients
	float4 D=computeLuebbersDiffractionCoefficient(k,n, angles.z,angles.w,angles.y,L, R_0, R_n);


	//Test: store diffraction coefficient
	//		RDNHit aHit;
	//		aHit.EEx=make_float4(0.0f,0.0f,0.0f,0.0f);
	//		aHit.EyEz=D;
	//		difBuffer[index]=aHit;
	//		return;
	//Split for better readability
	const float2 Ds=make_float2(D.x,D.y);
	const float2 Dh=make_float2(D.z,D.w);


	//Attenuation term at receiver due to propagation 
	z = make_float2(0.0f, -k*s);
	zexp = complex_exp_only_imaginary(z);
	float2 E_at_r = sca_complex_prod(-1.0f*A, zexp);

	//Some tests	
	//float2 Vbi=computeViB(zexp,Ds,Dh,s); 
	//float2 Vbr=computeVrB(zexp,Ds,Dh,s); 
	//float2 Vbir=Vbi+Vbr;
	//float2 Dhb=sca_complex_prod(-1.0*sqrtf(s),complex_prod(zexp,Vbir));
	//
	//	RDNHit aHit;
	//	aHit.EEx=make_float4(Vbi.x,Vbi.y, 0.0f,0.0f);
	//	aHit.EyEz=make_float4(0.0f,0.0f,0.0f,0.0f);
	//	difBuffer[index]=aHit;
	//return;
	//Complex amplitude of the diffracted E field at the receiver. Eq. [13-88] Balanis
	float2 Er_beta=complex_prod(E_at_r,complex_prod(Ds,Ei_beta_q)); //component of the diffracted E field parallel to the plane of diffraction at the receiver
	float2 Er_phi=complex_prod(E_at_r,complex_prod(Dh,Ei_phi_q)); //component of the diffracted E field perpendicular to the plane of diffraction at the  receiver
		//rtPrintf("%u \t Er_beta=(%6e,%6e) |Er_beta|=%6e  Er_phi=(%6e,%6e) |Er_phi|=%6e Ds=(%6e,%6e) Dh=(%6e,%6e))\n",launchIndex.y,Er_beta.x,Er_beta.y,length(Er_beta),Er_phi.x,Er_phi.y,length(Er_phi),D.x,D.y,D.z,D.w);


	//float2 Ee=Er_beta+Er_phi;

		//rtPrintf("%u \t Dhb=(%6e,%6e) Dh=(%6e,%6e)) angle(Vi+Vr)=%f angle(Er_phi)=%f a(E)=%f\n",launchIndex.y,Dhb.x,Dhb.y,Dh.x,Dh.y, (atan2(Vbir.y,Vbir.x)*180/M_PIf),(atan2(-Er_phi.y,-Er_phi.x)*180/M_PIf),(atan2(-Ee.y,-Ee.x)*180/M_PIf));

	//Get the unit vectors for the plane of diffraction. The above complex amplitude multiply the corresponding (beta and phi) unit vectors in the diffraction plane
	const float3 phi_u = n_dplane; 
	//s_unit vector is defined from DP to receiver, so we have to reverse ray here
	const float3 beta_o_u=normalize(cross(-rxRay,phi_u)); 
	//const float3 sss = normalize(cross(phi_u,beta_o_u));
		//rtPrintf("%u\t beta_o_u=(%f,%f,%f) phi_u=(%f,%f,%f) s=(%f,%f,%f) \n",launchIndex.x,beta_o_u.x,beta_o_u.y,beta_o_u.z,phi_u.x,phi_u.y,phi_u.z,sss.x,sss.y,sss.z);

	//Compute FIELD
		float3 xu=make_float3(1.0f,0.0f,0.0f);
		float3 yu=make_float3(0.0f,1.0f,0.0f);
		float3 zu=make_float3(0.0f,0.0f,1.0f);
		float2 Ex=sca_complex_prod(dot(beta_o_u,xu),Er_beta) + sca_complex_prod(dot(phi_u,xu),Er_phi);
		float2 Ey=sca_complex_prod(dot(beta_o_u,yu),Er_beta) + sca_complex_prod(dot(phi_u,yu),Er_phi);
		float2 Ez=sca_complex_prod(dot(beta_o_u,zu),Er_beta) + sca_complex_prod(dot(phi_u,zu),Er_phi);
			//printf("%u\t Ex=(%f,%f) |Ex|=%f Ey=(%f,%f) |Ey|=%f Ez=(%f,%f) |Ez|=%f \n",launchIndex.x,Ex.x, Ex.y,length(Ex),Ey.x,Ey.y,length(Ey),Ez.x,Ez.y,length(Ez));
		//float4 sangles=angles*57.2968f;
			//printf("%u\t%u\t%u  sangles(beta, beta',phi, phi')=(%f,%f,%f,%f) dif=%6e \n",launchIndex.x,launchIndex.y,launchIndex.z, sangles.x,sangles.y,sangles.z,sangles.w, (sangles.x-sangles.y));
		RDNHit aHit;
		float g=1;
		if (useAntennaGain) {
		
			g=getAntennaGain(rxRay, antennaGainIdBuffer[rxIndex],transformToPolarizationBuffer[rxIndex]);	
			Ex=sca_complex_prod(g,Ex);
			Ey=sca_complex_prod(g,Ey);
			Ez=sca_complex_prod(g,Ez);
			//printf("%u\t HDIF Ex=(%f,%f) |Ex|=%f Ey=(%f,%f) |Ey|=%f Ez=(%f,%f) |Ez|=%f g=%f \n",launchIndex.x,Ex.x, Ex.y,length(Ex),Ey.x,Ey.y,length(Ey),Ez.x,Ez.y,length(Ez),g);
		}

		//aHit.EEx=make_float4(0.0f,0.0f,Ex.x,Ex.y);
		//Additional output
		//float unfoldedPathLength = s+s_prime;
		//aHit.doaD = make_float4(rxRay.x, rxRay.y,rxRay.z, unfoldedPathLength);
		//aHit.doDu = make_float4(txRay.x, txRay.y,txRay.z, s);
	
		//difBuffer[index]=aHit;
	
	//Compute VOLTAGE

		//****************************
		//To get the induced voltage, we need to 
		//apply the dot product with the effective lenght of the received antenna. 

		//float3 ver_o; //Receiver vertical field vector
		//float3 hor_o; //Receiver horizontal field vector

		//Get polarization for receiver for this ray rxRay is already in the direction receiver to DP 
		//getLinearPolarizationInRayBasis(pol, rxRay,  hor_o,ver_o);

		//Get the  components of received field for the normal and parallel field vectors (geometric projection on receiver polarization vectors times reflection coefficients)
		//This would be equivalent to a dot product with the effective length (not the conjugated beacuse we already reversed and it is a linear polarization anyway)
		//const float2 Einorm=sca_complex_prod(dot(beta_o_u,hor_o),Er_beta) + sca_complex_prod(dot(phi_u,hor_o),Er_phi);
		//const float2 Eipar=sca_complex_prod(dot(beta_o_u,ver_o),Er_beta) + sca_complex_prod(dot(phi_u,ver_o),Er_phi);
		//const float2 Einorm=sca_complex_prod(dot(beta_o_u,beta_p),Er_beta) + sca_complex_prod(dot(phi_u,beta_p),Er_phi);
		//const float2 Eipar=sca_complex_prod(dot(beta_o_u,phi_p),Er_beta) + sca_complex_prod(dot(phi_u,phi_p),Er_phi);
		//float2 E=Einorm+Eipar;

		//The above formulation is equivalent to this below 
		//Geometric part due to polarization at the receiver
		//Get polarization for receiver for this ray rxRay is already in the direction receiver to DP 
		//const float3 Er_pol=getLinearPolarizationForRaySimple(pol,rxRay); //It is normalized
		const float3 Er_pol=getLinearPolarizationForRaySimple(receiverPolarizationBuffer[rxIndex],rxRay); //It is normalized

			//rtPrintf("%u\t%u\t%u\trxRay=(%f,%f,%f) Er_pol=(%f,%f,%f) Ev=(%f,%f,%f) A=%f )\n",launchIndex.x, launchIndex.y,launchIndex.z,rxRay.x,rxRay.y,rxRay.z,Er_pol.x,Er_pol.y,Er_pol.z,Ev.x,Ev.y,Ev.z,A);

		const float Er_beta_v=dot(beta_o_u,Er_pol); 
		const float Er_phi_v=dot(phi_u,Er_pol); 
		Er_beta=sca_complex_prod(Er_beta_v,Er_beta);
		Er_phi=sca_complex_prod(Er_phi_v,Er_phi);
		//This is actually the induced voltage on the antenna. From it we can compute the received power
		float2 E=Er_beta+Er_phi;
		if (useAntennaGain) {

			E=sca_complex_prod(g,E);
		}
		//float4 sangles=angles*57.2968f;

			//rtPrintf("%u\t%u\t%u  E=(%f,%f) E_b=(%f,%f) E_phi=(%f,%f) L=%f dif=%6e  \n",launchIndex.x,launchIndex.y,launchIndex.z, E.x,E.y,Er_beta.x,Er_beta.y,Er_phi.x, Er_phi.y, L, (sangles.x-sangles.y));
			//rtPrintf("G\t |E|=%6e index=(%u,%u,%u) %f \n",length(E),  index.x,index.y,index.z,angles.z*57.2968f);

			//rtPrintf("%u\t%u\t%u  sangles(beta, beta',phi, phi')=(%f,%f,%f,%f) L=%f dif=%6e  \n",launchIndex.x,launchIndex.y,launchIndex.z, sangles.x,sangles.y,sangles.z,sangles.w, L, (sangles.x-sangles.y));
		//RDNHit aHit;
		//aHit.EEx=make_float4(E.x,E.y, 0.0f,0.0f);
		aHit.EEx=make_float4(E.x,E.y,Ex.x,Ex.y); //Use 1 on EEx.x as flag for real hit
		aHit.EyEz=make_float4(Ey.x,Ey.y,Ez.x,Ez.y);
		//Additional output
		float unfoldedPathLength = s+s_prime;
		aHit.doaD = make_float4(rxRay.x, rxRay.y,rxRay.z, unfoldedPathLength);
		aHit.doDu = make_float4(txRay.x, txRay.y,txRay.z, s);
	
		difBuffer[launchIndex]=aHit;
		

}


//Diffraction Launch program
RT_PROGRAM void computeSingleDiffraction() {

	//1D launch [number_or_real_hist]
		//rtPrintf("%u\t%u\t%u Launch \n",launchIndex.x,launchIndex.y,launchIndex.z);
	const uint difBufferIndex=launchIndex;
	//Initialize buffer to make sure it does not carry values from previous launches
	RDNHit initHit;
	initHit.EEx=make_float4(0.0f,0.0f, 0.0f,0.0f);
	
	initHit.EyEz=make_float4(0.0f,0.0f,0.0f,0.0f);
	difBuffer[difBufferIndex]=initHit;
	const uint3 rhi=losBuffer[difBufferIndex];

	const uint edgeIndex=	rhi.x;
	const uint rxIndex = rhi.y;
	const uint txIndex = rhi.z;

	Transmitter tx = txBuffer[txIndex];

	const float3 origin = make_float3(tx.origin_p);
	const float4 sphere = receiverPositionsBuffer[rxIndex];
	//Check if ray is hitting his own tx (transmitter are also receivers usually) A transmitter cannot receive while it is transmitting, unless other channel is used.
	if (static_cast<int>(sphere.w)==tx.externalId) {
		return;	
	}	
	const float3 destination =make_float3(sphere.x,sphere.y,sphere.z);
	Edge e = edgeBuffer[edgeIndex];
	if (!isDiffractingEdge(origin,e)) {
		//rtPrintf("%u\t%u\t%u Not diffracting edge %d\n",launchIndex.x,launchIndex.y,launchIndex.z,e.id);
		return;
	}

	//Compute diffraction point (DP) between transmitter, receiver and edge
	float3 dp; 
	if (computeDiffractionPoint(origin,destination,e,dp)){
		//rtPrintf("%u\t%u\t%u e=%u dp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id,dp.x,dp.y,dp.z);
		float3 originToDP=dp-origin;
		float dist_originToDp=length(originToDP);
		float3 txRayDirection = originToDP/dist_originToDp;
		float3 destinationToDP=dp-destination;
		//rtPrintf("%u\t%u\t%u e=%u destinationToDP=(%f,%f,%f) tx not blocked \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, destinationToDP.x,destinationToDP.y,destinationToDP.z);
		float dist_destinationToDp=length(destinationToDP);
		float3 rxRayDir=destinationToDP/dist_destinationToDp;
		float3 rxRay=rxRayDir;
		//We are not going to trac, but keep it for backward compatibility
		VisibilityPayload visibilityRayPayload;
		visibilityRayPayload.polarization_k = tx.polarization_k; 
		visibilityRayPayload.result.x=OPAL_DIFFRACTION_LOS;
		visibilityRayPayload.faces=e.faces; //Faces of the edge, where the ray can hit to compute the diffraction. If it hits any other face, there is no LoS
		visibilityRayPayload.result.y=0;
		//LoS, so compute diffraction
		float2 spar; //Distance parameters [s,s']
		float3 n_iplane; //Normal vector of the incidence plane
		float3 n_dplane; //Normal vector of the diffraction plane
		float4 R_0; 
		float4 R_n;
		float4 angles=getDiffractionParameters<VisibilityPayload>(visibilityRayPayload,origin, destination,e,dp,spar, n_iplane, n_dplane, R_0, R_n);
		float4 sangles = angles*180.0f/M_PIf;

		//rtPrintf("%u\t%u\t%u e=%u  sangles(beta, beta',phi, phi')=(%f,%f,%f,%f) spar(s,s')=(%f,%f)   \n",launchIndex.x,(launchIndex.y),launchIndex.z,e.id, sangles.x,sangles.y,sangles.z,sangles.w,spar.x, spar.y);
		if (angles.z>=(M_PIf*e.pn.w)) {
			//Receiver is between face_0 and face_n (inside the edge). It cannot receive even if there is no blocking (actually blocking may not be detected by visibility)
			//rtPrintf("%u\t%u\t%u e=%u receiver is inside the wedge  sangles(beta, beta',phi, phi')=(%f,%f,%f,%f) spar(s,s')=(%f,%f)   \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, sangles.x,sangles.y,sangles.z,sangles.w,spar.x);
			return;
		}
		//rtPrintf("%u\t%u\t%u e=%u dp=(%f,%f,%f) rx not blocked \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, dp.x,dp.y,dp.z);
		float gain;
		if (useAntennaGain) {
			//Tx gain here. Rx gain in included in computeElectricFieldAtReceiver
			//rtPrintf("tx useAntennaGain\n");
			rtBufferId<float,2> bid=tx.gainId;
			const Matrix<4,4> tp=tx.transformToPolarization;
			gain=getAntennaGain(txRayDirection, bid, tp);
			//printf("%u\t%u\t%u gain=%f\n",launchIndex.x,launchIndex.y,launchIndex.z,gain) ; 
		} else {
			gain=1.0f;
		}
		computeElectricFieldAtReceiver(angles, spar, txRayDirection, tx.polarization_k,n_iplane,n_dplane,e.pn.w, rxRayDir, R_0, R_n, gain);
	} else {
		//rtPrintf("%u\t%u\t%u Not diffracting point on edge %d\n",launchIndex.x,launchIndex.y,launchIndex.z,e.id);
	}
}
//Closest hit program for triangles and visibility rays
//It is recommended to use CH instead of AH since AH forces to use the SM, see https://developer.download.nvidia.com/video/gputechconf/gtc/2019/presentation/s9768-new-features-in-optix-6.pdf
RT_PROGRAM void closestHitTriangleDiffraction() {
	//float3 hp=ray_hit.origin + ray_hit.direction*ch_triangle_data.geom_normal_t.w;
	//float3 hp=ray_hit.origin + ray_hit.direction*rtIntersectionDistance();
	//	rtPrintf("%u\t%u\t%u hit on face=%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,ch_triangle_data.faceId,hp.x,hp.y,hp.z);
	//rtPrintf("%u\t%u\t%u\t%d   hit on face=%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,ch_triangle_data.faceId,hp.x,hp.y,hp.z);
	if (ch_triangle_data.faceId!=rayPayload.faces.x && ch_triangle_data.faceId!=rayPayload.faces.y) { 
		//rtPrintf("%u\t%u\t%u\t%d blocked on face=%u ray.faces=%u,%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,ch_triangle_data.faceId,rayPayload.faces.x, rayPayload.faces.y,hp.x,hp.y,hp.z);
		//LoS is blocked, set flag 
		rayPayload.result.x=OPAL_DIFFRACTION_BLOCKED;
	} else {
		//rtPrintf("%u\t%u\t%u\t%d LoS on face=%u ray.faces=%u,%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,ch_triangle_data.faceId,rayPayload.faces.x, rayPayload.faces.y,hp.x,hp.y,hp.z);
	}

}
RT_PROGRAM void closestHitCurvedTriangleDiffraction() {
	//float3 hp=ray_hit.origin + ray_hit.direction*curved_triangle_data.geom_normal_t.w;
	if (curved_triangle_data.faceId!=rayPayload.faces.x && curved_triangle_data.faceId!=rayPayload.faces.y) { 
		//rtPrintf("%u\t%u\t%u\t%d blocked on curved face=%u ray.faces=%u,%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,curved_triangle_data.faceId,rayPayload.faces.x, rayPayload.faces.y,hp.x,hp.y,hp.z);
		//LoS is blocked, set flag 
		rayPayload.result.x=OPAL_DIFFRACTION_BLOCKED;
	} else {
		//rtPrintf("%u\t%u\t%u\t%d LoS on curved face=%u ray.faces=%u,%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,curved_triangle_data.faceId,rayPayload.faces.x, rayPayload.faces.y,hp.x,hp.y,hp.z);
	}

}
//Miss program for visibility rays
RT_PROGRAM void missDiffraction() {
	//rtPrintf("%u\t%u\t%u miss \n", launchIndex.x,launchIndex.y,launchIndex.z); 
	//rtPrintf("%u\t%u\t%u\t%d miss \n", launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y); 
	rayPayload.result.x=OPAL_DIFFRACTION_MISS;
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	if (RT_EXCEPTION_USER <= code)
	{
		printf("Diffraction computation user exception %d at (%d, %d)\n", code - RT_EXCEPTION_USER, launchIndex);
	}
	else
	{
		printf("Diffraction computation Exception code 0x%X at (%d, %d)\n", code, launchIndex);
	}

}

