#include "hip/hip_runtime.h"
/***************************************************************/
//
//Copyright (c) 2019 Esteban Egea-Lopez http://girtel.upct.es/~simulations
//
/**************************************************************/
#include "../../../Common.h"
#include <optix_world.h>
#include "../linearPolarizationFunctions.h"
#include "../../configuration.h"
#include "../../reflectionFunctions.h"
#include "../../receiverFunctions.h"
#include "diffractionFunctions.h"
using namespace optix;

//Constants
#define D_2_Pi 6.283185307179586f
#define D_SQRT_2_Pi 2.506628274631f
#define D_SQRT_2ByPi 0.797884560802865

//Scene root
//rtDeclareVariable(rtObject, root, , ); //Already defined in trace functions
//Static meshes root
//rtDeclareVariable(rtObject, staticMeshesRoot, , );

//Edge buffer
rtBuffer<Edge, 1> edgeBuffer;

//TODO:These  should be redundant if we used Receiver buffers
//Receiver position buffer 
rtBuffer<float4, 1> receiverPositionsBuffer;
//rtBuffer<float3, 1> receiverPolarizationBuffer;
//rtBuffer<rtBufferId<float,2>, 1> antennaGainIdBuffer;
//typedef optix::Matrix<4,4> TransMat; 
//rtBuffer<TransMat, 1> transformToPolarizationBuffer;


//Visibility buffer
rtBuffer<uint, 3> visibilityBuffer; //Buffer to store all the hits
//Hit buffer
//rtBuffer<RDNHit, 3> difBuffer; //Buffer to store all the hits

//Local variables
//rtDeclareVariable(float, k, , ); //wavenumber 2pi/lambda
//rtDeclareVariable(uint, computeMode, ,);
//rtDeclareVariable(uint, traceDiffraction, ,);

//Transmitter buffer
rtBuffer<Transmitter, 1> txBuffer;

//Launch variables
rtDeclareVariable(uint3, launchIndex, rtLaunchIndex, );

//Visibility ray payload
rtDeclareVariable(VisibilityPayload, rayPayload, rtPayload, );
rtDeclareVariable(uint, rayTypeIndex, , );

//For debug only
rtDeclareVariable(TriangleHit, ch_triangle_data, attribute triangle_hit_data, );
rtDeclareVariable(CurvedTriangleHit, curved_triangle_data, attribute curved_triangle_hit_data, );
rtDeclareVariable(optix::Ray, ray_hit, rtCurrentRay, );


//Diffraction Launch program
RT_PROGRAM void computeDiffractionVisibility() {

	//3D launch [edges,receivers,transmitters]
		//rtPrintf("%u\t%u\t%u Launch \n",launchIndex.x,launchIndex.y,launchIndex.z);
	uint3 difBufferIndex=launchIndex;
//	//Initialize buffer to make sure it does not carry values from previous launches
//	RDNHit aHit;
//	aHit.EEx=make_float4(0.0f,0.0f, 0.0f,0.0f);
//	aHit.EyEz=make_float4(0.0f,0.0f,0.0f,0.0f);
//	difBuffer[difBufferIndex]=aHit;
//
	visibilityBuffer[difBufferIndex]=0u;
	Transmitter tx = txBuffer[launchIndex.z];

	const float3 origin = make_float3(tx.origin_p);
	const float4 sphere = receiverPositionsBuffer[launchIndex.y];
	//Check if ray is hitting his own tx (transmitter are also receivers usually) A transmitter cannot receive while it is transmitting, unless other channel is used.
	if (static_cast<int>(sphere.w)==tx.externalId) {
		return;	
	}	
	const float3 destination =make_float3(sphere.x,sphere.y,sphere.z);
	Edge e = edgeBuffer[launchIndex.x];
	if (!isDiffractingEdge(origin,e)) {
		//rtPrintf("%u\t%u\t%u Not diffracting edge %d\n",launchIndex.x,launchIndex.y,launchIndex.z,e.id);
		return;
	}

	//Compute diffraction point (DP) between transmitter, receiver and edge
	float3 dp; 
	if (computeDiffractionPoint(origin,destination,e,dp)){
			//rtPrintf("%u\t%u\t%u e=%u dp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id,dp.x,dp.y,dp.z);
		VisibilityPayload visibilityRayPayload;
		visibilityRayPayload.polarization_k = tx.polarization_k; 
		visibilityRayPayload.result.x=OPAL_DIFFRACTION_LOS;
		visibilityRayPayload.faces=e.faces; //Faces of the edge, where the ray can hit to compute the diffraction. If it hits any other face, there is no LoS
		visibilityRayPayload.result.y=0;
		////trace visibility from transmitter to DP
		float3 originToDP=dp-origin;
		float dist_originToDp=length(originToDP);
		float3 txRayDirection = originToDP/dist_originToDp;
		//optix::Ray visibilityRay(origin,txRayDirection , rayTypeIndex, 0.0f,dist_originToDp); //Visibility ray type = 1
		optix::Ray visibilityRay(origin,txRayDirection , rayTypeIndex, min_t_epsilon,dist_originToDp-min_t_epsilon); //Visibility ray type = 1
			//rtPrintf("%u\t%u\t%u e=%u dp=(%f,%f,%f) tx ray=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, dp.x,dp.y,dp.z,visibilityRay.direction.x,visibilityRay.direction.y,visibilityRay.direction.z);
		//TODO: Only check visibility with static meshes so far. Change if we want to consider  moving meshes (such as vehicles)
		//WARNING: ONLY THIS METHODS WORKS. ANY OF THE ONE BELOW GIVES WRONG RESULTS, IT MAY BE A OPTIX BU
		rtTrace(root, visibilityRay, visibilityRayPayload,OPAL_STATIC_MESH_MASK,RT_RAY_FLAG_DISABLE_ANYHIT);

		//rtTrace(root, visibilityRay, visibilityRayPayload,RT_VISIBILITY_ALL);
		//rtTrace(root, visibilityRay, visibilityRayPayload,RT_VISIBILITY_ALL,RT_RAY_FLAG_DISABLE_ANYHIT);
		//rtTrace(root, visibilityRay, visibilityRayPayload);
		if (visibilityRayPayload.result.x!=OPAL_DIFFRACTION_BLOCKED) {
				//rtPrintf("%u\t%u\t%u e=%u dp=(%f,%f,%f) tx not blocked \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, dp.x,dp.y,dp.z);
			//trace visibility from receiver to DP
			float3 destinationToDP=dp-destination;
				//rtPrintf("%u\t%u\t%u e=%u destinationToDP=(%f,%f,%f) tx not blocked \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, destinationToDP.x,destinationToDP.y,destinationToDP.z);
			float dist_destinationToDp=length(destinationToDP);
			visibilityRay.origin=destination;
			visibilityRay.direction=destinationToDP/dist_destinationToDp;
			visibilityRay.tmin=0.0f;
			visibilityRay.tmax=dist_destinationToDp; 
			float3 rxRayDir=destinationToDP/dist_destinationToDp;
			//optix::Ray visibilityRayRx(destination, rxRayDir , rayTypeIndex, 0.0f,dist_destinationToDp); //Visibility ray type = 1
			optix::Ray visibilityRayRx(destination, rxRayDir , rayTypeIndex, min_t_epsilon,dist_destinationToDp-min_t_epsilon); //Visibility ray type = 1

			VisibilityPayload visibilityRayPayloadRx;
			visibilityRayPayloadRx.polarization_k = tx.polarization_k; 
			visibilityRayPayloadRx.result.x=OPAL_DIFFRACTION_LOS;
			visibilityRayPayloadRx.faces=e.faces; //Faces of the edge, where the ray can hit to compute the diffraction. If it hits any other face, there is no LoS
			visibilityRayPayloadRx.result.y=1;
			//visibilityRayPayload.result.x=OPAL_DIFFRACTION_LOS;
			//visibilityRayPayload.result.y=1;
				//rtPrintf("%u\t%u\t%u e=%u dp=(%f,%f,%f) rx ray=(%f,%f,%f) d=%f \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, dp.x,dp.y,dp.z,visibilityRay.direction.x,visibilityRay.direction.y,visibilityRay.direction.z, dist_destinationToDp	);
			float3 rxRay=visibilityRay.direction;
				//rtPrintf("%u\t%u\t%u e=%u rxRay=(%f,%f,%f) rx not blocked \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, rxRay.x,rxRay.y,rxRay.z);
				//rtPrintf("%u\t%u\t%u e=%u diff sangles(beta, beta',phi, phi')=(%f,%f,%f,%f) spar(s,s')=(%f,%f) face_0=(%f,%f,%f)  \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, sangles.x,sangles.y,sangles.z,sangles.w,spar.x,spar.y, face_0.x,face_0.y,face_0.z);

			rtTrace(root, visibilityRayRx, visibilityRayPayloadRx,OPAL_STATIC_MESH_MASK,RT_RAY_FLAG_DISABLE_ANYHIT);
			//rtTrace(root, visibilityRayRx, visibilityRayPayload,RT_VISIBILITY_ALL,RT_RAY_FLAG_DISABLE_ANYHIT);
			//rtTrace(root, visibilityRay, visibilityRayPayload,RT_VISIBILITY_ALL);
			//rtTrace(root, visibilityRayRx, visibilityRayPayloadRx);
			if (visibilityRayPayloadRx.result.x!=OPAL_DIFFRACTION_BLOCKED) {
				//LoS, so compute diffraction
				float2 spar; //Distance parameters [s,s']
				float3 n_iplane; //Normal vector of the incidence plane
				float3 n_dplane; //Normal vector of the diffraction plane
				float4 R_0; 
				float4 R_n;
				float4 angles=getDiffractionParameters<VisibilityPayload>(visibilityRayPayload,origin, destination,e,dp,spar, n_iplane, n_dplane, R_0, R_n);
				float4 sangles = angles*180.0f/M_PIf;
				
					//rtPrintf("%u\t%u\t%u e=%u  sangles(beta, beta',phi, phi')=(%f,%f,%f,%f) spar(s,s')=(%f,%f)   \n",launchIndex.x,(launchIndex.y),launchIndex.z,e.id, sangles.x,sangles.y,sangles.z,sangles.w,spar.x, spar.y);
				if (angles.z>=(M_PIf*e.pn.w)) {
					//Receiver is between face_0 and face_n (inside the edge). It cannot receive even if there is no blocking (actually blocking may not be detected by visibility)
						//rtPrintf("%u\t%u\t%u e=%u receiver is inside the wedge  sangles(beta, beta',phi, phi')=(%f,%f,%f,%f) spar(s,s')=(%f,%f)   \n",launchIndex.x,launchIndex.y,launchIndex.z,e.id, sangles.x,sangles.y,sangles.z,sangles.w,spar.x);
					return;
				}
				visibilityBuffer[difBufferIndex]=1;
			} 
		}
	} else {
		//rtPrintf("%u\t%u\t%u Not diffracting point on edge %d\n",launchIndex.x,launchIndex.y,launchIndex.z,e.id);
	}
}
//Closest hit program for triangles and visibility rays
//It is recommended to use CH instead of AH since AH forces to use the SM, see https://developer.download.nvidia.com/video/gputechconf/gtc/2019/presentation/s9768-new-features-in-optix-6.pdf
RT_PROGRAM void closestHitTriangleDiffraction() {
	//float3 hp=ray_hit.origin + ray_hit.direction*ch_triangle_data.geom_normal_t.w;
	//float3 hp=ray_hit.origin + ray_hit.direction*rtIntersectionDistance();
	//	rtPrintf("%u\t%u\t%u hit on face=%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,ch_triangle_data.faceId,hp.x,hp.y,hp.z);
	//rtPrintf("%u\t%u\t%u\t%d   hit on face=%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,ch_triangle_data.faceId,hp.x,hp.y,hp.z);
	if (ch_triangle_data.faceId!=rayPayload.faces.x && ch_triangle_data.faceId!=rayPayload.faces.y) { 
		//rtPrintf("%u\t%u\t%u\t%d blocked on face=%u ray.faces=%u,%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,ch_triangle_data.faceId,rayPayload.faces.x, rayPayload.faces.y,hp.x,hp.y,hp.z);
		//LoS is blocked, set flag 
		rayPayload.result.x=OPAL_DIFFRACTION_BLOCKED;
	} else {
		//rtPrintf("%u\t%u\t%u\t%d LoS on face=%u ray.faces=%u,%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,ch_triangle_data.faceId,rayPayload.faces.x, rayPayload.faces.y,hp.x,hp.y,hp.z);
	}

}
RT_PROGRAM void closestHitCurvedTriangleDiffraction() {
	//float3 hp=ray_hit.origin + ray_hit.direction*curved_triangle_data.geom_normal_t.w;
	if (curved_triangle_data.faceId!=rayPayload.faces.x && curved_triangle_data.faceId!=rayPayload.faces.y) { 
		//rtPrintf("%u\t%u\t%u\t%d blocked on curved face=%u ray.faces=%u,%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,curved_triangle_data.faceId,rayPayload.faces.x, rayPayload.faces.y,hp.x,hp.y,hp.z);
		//LoS is blocked, set flag 
		rayPayload.result.x=OPAL_DIFFRACTION_BLOCKED;
	} else {
		//rtPrintf("%u\t%u\t%u\t%d LoS on curved face=%u ray.faces=%u,%u hp=(%f,%f,%f) \n",launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y,curved_triangle_data.faceId,rayPayload.faces.x, rayPayload.faces.y,hp.x,hp.y,hp.z);
	}

}
//Miss program for visibility rays
RT_PROGRAM void missDiffraction() {
	//rtPrintf("%u\t%u\t%u miss \n", launchIndex.x,launchIndex.y,launchIndex.z); 
	//rtPrintf("%u\t%u\t%u\t%d miss \n", launchIndex.x,launchIndex.y,launchIndex.z,rayPayload.result.y); 
	rayPayload.result.x=OPAL_DIFFRACTION_MISS;
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	if (RT_EXCEPTION_USER <= code)
	{
		printf("Diffraction visibility user exception %d at (%d, %d)\n", code - RT_EXCEPTION_USER, launchIndex.x, launchIndex.y);
	}
	else
	{
		printf("Diffraction visibility Exception code 0x%X at (%d, %d)\n", code, launchIndex.x, launchIndex.y);
	}

}

